#include <stdio.h>
#include <stdlib.h>
#include <string.h>
#include <assert.h>
#include <hip/hip_runtime.h>
#include <unistd.h>
#include <sys/time.h>
#include <sys/mman.h>
#include <unistd.h>
#include <fcntl.h>

#define CEIL(a, b) ( ((a) + (b) - 1) / (b) )
#define MIN(a, b) ( (a) < (b) ? (a) : (b) )
#define CAL_TIME ( 1e-6 * (temp_time.tv_usec - start_time.tv_usec) + (temp_time.tv_sec - start_time.tv_sec) )
#define C2I(i) ( ptr[i] - '0')
#define ROW_COL(__i) ( __i / line_d ), ( ( __i % pitch ) / block_size )


const int INF = 1000000000;
const int V = 20010;
const int block_size = 32;
int max_streams = 4;
int first_round = 4;
dim3 threads(block_size, block_size);

void input(char *outFileName);
void block_FW();
void block_FW_S();
void split_strings(char *ptr);
void cuda_init();
void cuda_cleanup();

__constant__ unsigned int *Dist;
__constant__ int pitch_d;

__global__ void cal_phase1(int pivot)
{
    __shared__ unsigned int block_dist[block_size][block_size];

    int tx = threadIdx.x, ty = threadIdx.y;
    int tid = ty * pitch_d + tx;
    int block_index = pivot + tid;
    unsigned int origin, blk_dist, new_dist;

    block_dist[ty][tx] = origin = Dist[block_index];
    __syncthreads();

    if(origin > INF)
        Dist[block_index] = origin = INF;

    blk_dist = origin;
    for(int k=0; k<block_size-1; k++) {
        new_dist = block_dist[ty][k] + block_dist[k][tx];
        //if (block_dist[ty][tx] > new_dist)
            //block_dist[ty][tx] = new_dist;
        if(blk_dist > new_dist)
            block_dist[ty][tx] = blk_dist = new_dist;
        __syncthreads();
    }
    new_dist = block_dist[ty][block_size-1] + block_dist[block_size-1][tx];
    if(blk_dist > new_dist)
        Dist[block_index] = new_dist;
    else if(origin > blk_dist)
        Dist[block_index] = blk_dist;
}

__global__ void cal_phase2_row(int pivot, int r)
{
    __shared__ unsigned int block_dist[block_size][block_size+1];
    __shared__ unsigned int pivot_dist[block_size][block_size+1];

    int tx = threadIdx.x, ty = threadIdx.y;
    int tid = ty * pitch_d + tx;
    int column = block_size * (blockIdx.x - r);
    int block_index, pivot_index;
    unsigned int blk_dist, new_dist, origin;

    pivot_index = pivot + tid;

    if(blockIdx.x==r) // pivot block
        return;
/*
    block_index = pivot_index + column;
    block_dist[ty][tx] = origin = Dist[block_index];
    __syncthreads();

    if(origin > INF)
        Dist[block_index] = origin = INF;

    pivot_dist[ty][tx] = Dist[pivot_index];

    blk_dist = origin;
    for(int k=0; k<block_size-1; k++) {
        new_dist = pivot_dist[ty][k] + block_dist[k][tx];

        //if (block_dist[ty][tx] > new_dist)
            //block_dist[ty][tx] = new_dist;
        if (blk_dist > new_dist)
            block_dist[ty][tx] = blk_dist = new_dist;
        __syncthreads();
    }
    new_dist = pivot_dist[ty][block_size-1] + block_dist[block_size-1][tx];

    if(blk_dist > new_dist)
        Dist[block_index] = new_dist;
    else if(origin > blk_dist)
        Dist[block_index] = blk_dist;
*/
    pivot_dist[ty][tx] = Dist[pivot_index];
    block_index = pivot_index + column;
    block_dist[tx][ty] = origin = Dist[block_index];
    __syncthreads();

    if(origin > INF)
        Dist[block_index] = origin = INF;

    blk_dist = block_dist[ty][tx];
    for(int k=0; k<block_size; k++) {
        new_dist = pivot_dist[tx][k] + block_dist[ty][k];

        if (blk_dist > new_dist)
            block_dist[ty][tx] = blk_dist = new_dist;
    }
    __syncthreads();

    blk_dist = block_dist[tx][ty];
    if(origin > blk_dist)
        Dist[block_index] = blk_dist;
}

__global__ void cal_phase2_blk(int p1_pivot, int p2_pivot)
{
    __shared__ unsigned int block_dist[block_size][block_size];
    __shared__ unsigned int pivot_dist[block_size][block_size];

    int tx = threadIdx.x, ty = threadIdx.y;
    int tid = ty * pitch_d + tx;
    int block_index;
    unsigned int origin, blk_dist, new_dist;

    pivot_dist[ty][tx] = Dist[p1_pivot + tid];
    __syncthreads();

    block_index = p2_pivot + tid + blockIdx.x * pitch_d * block_size;
    block_dist[ty][tx] = origin = Dist[block_index];

    blk_dist = origin;
    for(int k=0; k<block_size-1; k++) {
        new_dist = block_dist[ty][k] + pivot_dist[k][tx];

        if(blk_dist > new_dist)
            block_dist[ty][tx] = blk_dist = new_dist;
    }
    new_dist = block_dist[ty][block_size-1] + pivot_dist[block_size-1][tx];
    if(blk_dist > new_dist)
        Dist[block_index] = new_dist;
    else if(origin > blk_dist)
        Dist[block_index] = blk_dist;
}

__global__ void cal_phase3(int p1_pivot, int p2_pivot, int r)
{
    __shared__ unsigned int pvRow_dist[block_size][block_size];
    __shared__ unsigned int pvCol_dist[block_size][block_size];

    int tx = threadIdx.x, ty = threadIdx.y;
    int tid = ty * pitch_d + tx;
    int col_diff = (blockIdx.x - r) * block_size;
    int block_index, p1_index, p2_index;
    unsigned int origin, block_dist, new1, new2;

    p1_index = p1_pivot + col_diff + tid;
    p2_index = p2_pivot + tid;

    if(col_diff==0) // pivots
        return;

    pvRow_dist[ty][tx] = Dist[p1_index];
    pvCol_dist[ty][tx] = Dist[p2_index];
    __syncthreads();

    block_dist = pvCol_dist[ty][0] + pvRow_dist[0][tx];
    new1 = pvCol_dist[ty][1] + pvRow_dist[1][tx];

    block_index = p2_index + col_diff;
    origin = Dist[block_index];

    if (block_dist > new1)
        block_dist = new1;

    for(int k=2; k<block_size; k+=2) {
        new1 = pvCol_dist[ty][k] + pvRow_dist[k][tx];
        new2 = pvCol_dist[ty][k+1] + pvRow_dist[k+1][tx];
        if (block_dist > new1)
            block_dist = new1;
        if (block_dist > new2)
            block_dist = new2;
    }
    if(origin>block_dist)
        Dist[block_index] = block_dist;
    //Dist[block_index] = MIN(origin, block_dist);
}

__global__ void cal_phase3_n(int p1_pivot, int p2_pivot, int r, int n)
{
    __shared__ unsigned int pvR_dist[block_size][block_size];
    __shared__ unsigned int pvC_dist[2][block_size][block_size];

    int tx = threadIdx.x, ty = threadIdx.y;
    int tid = ty * pitch_d + tx;
    int col_diff = (blockIdx.x - r) * block_size;
    int row_diff = pitch_d * block_size;
    int b1_index, b2_index, p_index;
    unsigned int origin, b1_dist, b2_dist;
    unsigned int inter[block_size], new1, new2;
    int p1 = 0, p2 = 1;

    p_index = p1_pivot + tid + col_diff;

    if(col_diff==0) // pivots
        return;

    pvR_dist[ty][tx] = Dist[p_index];
    __syncthreads();
    for(int k=0; k<block_size; k++)
        inter[k] = pvR_dist[k][tx];

    p_index = p2_pivot + tid;
    pvC_dist[p1][ty][tx] = Dist[p_index];
    b1_index = p_index + col_diff;
    b1_dist = origin = Dist[b1_index];
    while(n-->1) {
        p_index += row_diff;
        pvC_dist[p2][ty][tx] = Dist[p_index];
        b2_index = b1_index + row_diff;
        b2_dist = Dist[b2_index];

        for(int k=0; k<block_size; k+=2) {
            new1 = pvC_dist[p1][ty][k] + inter[k];
            new2 = pvC_dist[p1][ty][k+1] + inter[k+1];
            if (b1_dist > new1)
                b1_dist = new1;
            if (b1_dist > new2)
                b1_dist = new2;
        }
        if (origin > b1_dist)
            Dist[b1_index] = b1_dist;
        //Dist[b1_index] = MIN(origin, b1_dist);

        p1 ^= 1;
        p2 ^= 1;
        b1_dist = origin = b2_dist;
        b1_index = b2_index;
    }
    for(int k=0; k<block_size; k+=2) {
        new1 = pvC_dist[p1][ty][k] + inter[k];
        new2 = pvC_dist[p1][ty][k+1] + inter[k+1];
        if (b1_dist > new1)
            b1_dist = new1;
        if (b1_dist > new2)
            b1_dist = new2;
    }
    if (origin > b1_dist)
        Dist[b1_index] = b1_dist;
    //Dist[b1_index] = MIN(origin, b1_dist);
}


int n, n_bytes, out_size; // Number of vertices, edges
int Rounds, b_rounds, b_rounds_bytes;
int line_n, last_line, max_row;
FILE *infile;
int out_fd;

struct timeval start_time, temp_time;

unsigned int *Dist_h, *Dist_d;
int pitch_bytes, pitch;
int diag_size, line_d_bytes, line_d;

hipStream_t stream[8], stream_s, stream_m;
hipEvent_t ev_1, ev_2, ev_m;


int main(int argc, char* argv[])
{
    assert(argc==4);
    //block_size = atoi(argv[3]);
    gettimeofday(&start_time, NULL);

    infile  = fopen(argv[1], "r");
    input(argv[2]);

    gettimeofday(&temp_time, NULL);
    //printf("input> %g s\n", CAL_TIME);

    if(Rounds<=8) {
        block_FW_S();
    } else {
        block_FW();
        //printf("NOP\n");
    }

    hipEventRecord(ev_m, stream_m);
    hipEventSynchronize(ev_m);
    msync(Dist_h, out_size, MS_SYNC);
    munmap(Dist_h, out_size);
    close(out_fd);

    cuda_cleanup();

    gettimeofday(&temp_time, NULL);
    //printf("block_FW> %g s\n", CAL_TIME);

    return 0;
}

void cuda_init()
{
    int bline = Rounds==1 ? n : block_size;
    hipMemcpy2DAsync(Dist_d, pitch_bytes, Dist_h, n_bytes, n_bytes, bline, hipMemcpyHostToDevice);

    hipStreamCreate(&stream_m);
    hipStreamCreate(&stream[0]);
    cal_phase1<<<1, threads, 0, stream[0]>>>(0);
    hipEventCreateWithFlags(&ev_1, hipEventDisableTiming);
    hipEventRecord(ev_1, stream[0]);

    cal_phase2_row<<<Rounds, threads, 0, stream[0]>>>(0, 0);
    hipEventCreateWithFlags(&ev_2, hipEventDisableTiming);
    hipEventRecord(ev_2, stream[0]);

    hipEventCreateWithFlags(&ev_m, hipEventDisableTiming);
}

void cuda_cleanup()
{
    hipDeviceSynchronize();
    int num_streams;
    if(Rounds<=8) {
        num_streams = Rounds;
    } else {
        num_streams = max_streams;
        hipStreamDestroy(stream_s);
    }
    hipStreamDestroy(stream_m);
    for(int i=0; i<num_streams; i++) {
        hipStreamDestroy(stream[i]);
    }
    hipEventDestroy(ev_1);
    hipEventDestroy(ev_2);
    hipEventDestroy(ev_m);
    hipFree(Dist_d);
}

void block_FW()
{
    int id_1[V], do_r[V], row;
    int p1_start = 0, p2_start = 0, p2_sub;
    unsigned int *ptr_h = Dist_h, *ptr_d = Dist_d;
    int flag, bline = block_size;
    hipStream_t *sp, s;

    cuda_init();

    id_1[0] = 0;
    do_r[0] = max_row;
    //printf("Round 1: row < first_round (in pivot)\n");
    for(int i=1; i<first_round; i++) {
        sp = &stream[i];
        id_1[i] = i / max_row;
        do_r[i] = max_row - i % max_row;
        ptr_h += line_n;
        ptr_d += line_d;
        hipMemcpy2DAsync(ptr_d, pitch_bytes, ptr_h, n_bytes, n_bytes, block_size, hipMemcpyHostToDevice, stream_m);
        hipEventRecord(ev_m, stream_m);

        hipStreamCreate(sp);
        s = *sp;
        hipStreamWaitEvent(s, ev_1, 0);
        hipStreamWaitEvent(s, ev_m, 0);

        p2_start += line_d;
        cal_phase2_blk<<< 1, threads, 0, s>>>(p1_start, p2_start);
        hipStreamWaitEvent(s, ev_2, 0);
        cal_phase3<<<Rounds, threads, 0, s>>>(p1_start, p2_start, 0);
    }

    //printf("Round (2-first_round): row < first_round\n");
    for(int i=1; i<first_round; i++) {
        s = stream[i];
        p1_start += diag_size;
        //printf("round %d: p1=(%d,%d) stream %d\n", i, ROW_COL(p1_start), i);
        cal_phase1<<<1, threads, 0, s>>>(p1_start);
        hipEventRecord(ev_1, s);
        cal_phase2_row<<<Rounds, threads, 0, s>>>(p1_start, i);
        hipEventRecord(ev_2, s);

        for(int j=0; j<first_round; j++) {
            if(i==j) continue;
            hipStream_t sj = stream[j];

            p2_sub = p1_start + line_d * (j - i);
            //printf("\tp1=(%d,%d), p2=(%d,%d) stream %d\n", ROW_COL(p1_start), ROW_COL(p2_sub), j);
            hipStreamWaitEvent(sj, ev_1, 0);
            cal_phase2_blk<<< 1, threads, 0, sj>>>(p1_start, p2_sub);
            hipStreamWaitEvent(sj, ev_2, 0);
            cal_phase3<<<Rounds, threads, 0, sj>>>(p1_start, p2_sub, i);
        }
    }

    for(int i=0; i<max_streams; i++) {
        hipEventRecord(ev_1, stream[i]);
        for(int j=0; j<max_streams; j++) {
            if(i==j) continue;
            hipStreamWaitEvent(stream[j], ev_1, 0);
        }
    }

    //printf("Round (1-first_round): other rows\n");
    flag = 1;
    for(int i=first_round; i<Rounds; i++) {
        id_1[i] = i / max_row % max_streams;
        do_r[i] = max_row - i % max_row;
        if(i + do_r[i] > Rounds)
            do_r[i] = Rounds - i;

        s = stream[id_1[i]];

        ptr_h += line_n;
        ptr_d += line_d;
        p2_start += line_d;
        if(flag>0) {
            if(i==first_round) {
                row = 1;
                bline = block_size;
            } else {
                row = do_r[i];
                bline = (i+do_r[i]==Rounds) ? last_line + (do_r[i]-1) * block_size : do_r[i] * block_size;
            }
            hipMemcpy2DAsync(ptr_d, pitch_bytes, ptr_h, n_bytes, n_bytes, bline, hipMemcpyHostToDevice, stream_m);
            hipEventRecord(ev_m, stream_m);
            hipStreamWaitEvent(s, ev_m, 0);

            p1_start = 0;
            p2_sub = p2_start;
            for(int r=0; r<first_round; r++) {

                cal_phase2_blk<<<row, threads, 0, s>>>(p1_start, p2_sub);
                cal_phase3_n<<<Rounds, threads, 0, s>>>(p1_start, p2_sub, r, row);
                p1_start += diag_size;
                p2_sub += block_size;
            }
            if(i==first_round) {
                hipStreamCreate(&stream_s);
                hipEventRecord(ev_1, s);
                hipStreamWaitEvent(stream_s, ev_1, 0);
            }
            flag -= row - 1;
        } else {
            flag++;
        }
    }

    //printf("R %d\n", Rounds);
    for (int r=first_round; r<Rounds; ++r) {
        cal_phase1<<<1, threads, 0, stream_s>>>(p1_start);
        hipEventRecord(ev_1, stream_s);
        cal_phase2_row<<<Rounds, threads, 0, stream_s>>>(p1_start, r);
        hipEventRecord(ev_2, stream_s);

        if(r==Rounds-1) {
            bline = last_line;
            ptr_h = Dist_h + r * line_n;
            ptr_d = Dist_d + r * line_d;
            hipStreamWaitEvent(stream_m, ev_2, 0);
            hipMemcpy2DAsync(ptr_h, n_bytes, ptr_d, pitch_bytes, n_bytes, bline, hipMemcpyDeviceToHost, stream_m);
            //cudaMemcpy2D(ptr_h, n_bytes, ptr_d, pitch_bytes, n_bytes, bline, cudaMemcpyDeviceToHost);
        }

        //printf("r %d\n", r);
        int next_r = r + 1;
        if(next_r<Rounds) {
            s = stream[id_1[next_r]];
            p2_start = p1_start + line_d;
            hipStreamWaitEvent(s, ev_1, 0);
            cal_phase2_blk<<<1, threads, 0, s>>>(p1_start, p2_start);
            hipStreamWaitEvent(s, ev_2, 0);
            cal_phase3<<<Rounds, threads, 0, s>>>(p1_start, p2_start, r);

            hipEventRecord(ev_m, s);
            hipStreamWaitEvent(stream_s, ev_m, 0);

        }

        flag = 1;
        for(int i = (r+1) % Rounds; i != r; i = (i==Rounds-1) ? 0 : i+1) {
            if(i==r+1) continue;

            s = stream[id_1[i]];
            p2_start = p1_start + line_d * (i-r);
            if(flag>0) {
                row = (i<r && i+do_r[i]>r) ? r - i : do_r[i];
                flag -= row - 1;

                hipStreamWaitEvent(s, ev_1, 0);
                cal_phase2_blk<<<row, threads, 0, s>>>(p1_start, p2_start);
                hipStreamWaitEvent(s, ev_2, 0);
                cal_phase3_n<<<Rounds, threads, 0, s>>>(p1_start, p2_start, r, row);
                
                if(r==Rounds-1) {
                    bline = row * block_size;
                    ptr_h = Dist_h + i * line_n;
                    ptr_d = Dist_d + i * line_d;

                    hipEventRecord(ev_m, s);
                    hipStreamWaitEvent(stream_m, ev_m, 0);

                    //cudaMemcpy2DAsync(ptr_h, n_bytes, ptr_d, pitch_bytes, n_bytes, bline, cudaMemcpyDeviceToHost, stream_m);
                    hipMemcpy2D(ptr_h, n_bytes, ptr_d, pitch_bytes, n_bytes, bline, hipMemcpyDeviceToHost);
                }
            } else {
                flag++;
            }
        }
        p1_start += diag_size;
    }
}

size_t m, sz;
void input(char *outFileName)
{
    char *tok_1, *tok_2, *fstr;
    char temp[30];
    size_t p_bytes;

    fseek(infile, 0L, SEEK_END);
    sz = ftell(infile);
    fseek(infile, 0L, SEEK_SET);

    fstr = (char *) mmap(NULL, sz, PROT_READ, MAP_PRIVATE|MAP_POPULATE, fileno(infile), 0);
    if(fstr==MAP_FAILED) {
        fprintf(stderr, "mmap faild fstr\n");
        exit(1);
    }

    tok_1 = strchr(fstr, ' ');
    strncpy(temp, fstr, tok_1-fstr);
    n = atoi(temp);
    tok_1++;
    tok_2 = strchr(tok_1, '\n');
    strncpy(temp, tok_1, tok_2-tok_1);
    m = atoi(temp);
    tok_2++;

    Rounds = CEIL(n, block_size);
    b_rounds = block_size * Rounds;
    b_rounds_bytes = b_rounds * sizeof(int);

    gettimeofday(&temp_time, NULL);
    //printf("before parsing> %g s\n", CAL_TIME);

    n_bytes = n * sizeof(int);
    last_line = n - (Rounds-1) * block_size;
    out_size = n * n_bytes;
    max_row = (Rounds+max_streams-1) / max_streams;

    int fflag = S_IRUSR | S_IWUSR | S_IRGRP | S_IWGRP | S_IROTH | S_IWOTH;
    out_fd = open(outFileName, O_RDWR|O_CREAT, fflag);
    if(0!=posix_fallocate(out_fd, 0, out_size)) {
        fprintf(stderr, "posix_fallocate failed\n");
        exit(1);
    }
    Dist_h = (unsigned int *) mmap(NULL, out_size, PROT_READ|PROT_WRITE, MAP_SHARED, out_fd, 0);
    if(Dist_h==MAP_FAILED) {
        fprintf(stderr, "mmap faild Dist_h\n");
        exit(1);
    }
    memset(Dist_h, 64, out_size);
    for (int i = 0; i < n*n; i+=n+1)
        Dist_h[i] = 0;
    //fprintf(stderr, "memset success\n");

    gettimeofday(&temp_time, NULL);
    //printf("\tfile read done> %g s\n", CAL_TIME);

    split_strings(tok_2);
    munmap(fstr, sz);
    fclose(infile);

    gettimeofday(&temp_time, NULL);
    //printf("\tparsing done> %g s\n", CAL_TIME);
    if(n>=10000) {
        max_streams = 6;
        first_round = 6;
    }
    hipMallocPitch(&Dist_d, &p_bytes, b_rounds_bytes, b_rounds);
    pitch_bytes = p_bytes;
    pitch = pitch_bytes / sizeof(int);
    hipMemcpyToSymbolAsync(HIP_SYMBOL(Dist), &Dist_d, sizeof(Dist_d), 0);
    hipMemcpyToSymbolAsync(HIP_SYMBOL(pitch_d), &pitch, sizeof(pitch), 0);
    hipMemset2DAsync(Dist_d, p_bytes, 64, b_rounds_bytes, b_rounds);
    line_n = block_size * n;
    line_d = block_size * pitch;
    diag_size = (pitch + 1) * block_size;

    fprintf(stderr, "n %d, Rounds %d, streams %d, rows %d\n", n, Rounds, max_streams, max_row);
    gettimeofday(&temp_time, NULL);
    //printf("\tcuda allocate done> %g s\n", CAL_TIME);
}

void split_strings(char *ptr)
{
    int a, b, v;
    while(m-->0) {
        if(ptr[1]==' ') {
            a = C2I(0);
            ptr += 2;
        } else if(ptr[2]==' ') {
            a = C2I(0) * 10 + C2I(1);
            ptr += 3;
        } else if(ptr[3]==' ') {
            a = C2I(0) * 100 + C2I(1) * 10 + C2I(2);
            ptr += 4;
        } else if(ptr[4]==' ') {
            a = C2I(0) * 1000 + C2I(1) * 100 + C2I(2) * 10 + C2I(3);
            ptr += 5;
        } else {
            a = C2I(0) * 10000 + C2I(1) * 1000 + C2I(2) * 100 + C2I(3) * 10 + C2I(4);
            ptr += 6;
        }

        if(ptr[1]==' ') {
            b = C2I(0);
            ptr += 2;
        } else if(ptr[2]==' ') {
            b = C2I(0) * 10 + C2I(1);
            ptr += 3;
        } else if(ptr[3]==' ') {
            b = C2I(0) * 100 + C2I(1) * 10 + C2I(2);
            ptr += 4;
        } else if(ptr[4]==' ') {
            b = C2I(0) * 1000 + C2I(1) * 100 + C2I(2) * 10 + C2I(3);
            ptr += 5;
        } else {
            b = C2I(0) * 10000 + C2I(1) * 1000 + C2I(2) * 100 + C2I(3) * 10 + C2I(4);
            ptr += 6;
        }

        if(ptr[1]=='\n') {
            v = C2I(0);
            ptr += 2;
        } else if(ptr[2]=='\n') {
            v = C2I(0) * 10 + C2I(1);
            ptr += 3;
        } else {
            v = C2I(0) * 100 + C2I(1) * 10 + C2I(2);
            ptr += 4;
        }

        Dist_h[ n * a + b ] = v;
    }
}

void block_FW_S()
{
    int p1_start = 0, p2_start = 0;
    unsigned int *ptr_h = Dist_h, *ptr_d = Dist_d;
    int p2_sub, bline;

    cuda_init();

    //printf("round 1\n");
    for(int i=1; i<Rounds; i++) {
        ptr_h += line_n;
        ptr_d += line_d;
        bline = i==Rounds-1 ? last_line : block_size;
        hipMemcpy2DAsync(ptr_d, pitch_bytes, ptr_h, n_bytes, n_bytes, bline, hipMemcpyHostToDevice, stream_m);
        hipEventRecord(ev_m, stream_m);

        hipStreamCreate(&stream[i]);
        hipStreamWaitEvent(stream[i], ev_m, 0);

        p2_start += line_d;
        //printf("\tp1=(%d,%d), p2=(%d,%d) stream %d\n", ROW_COL(p1_start), ROW_COL(p2_start), i);
        hipStreamWaitEvent(stream[i], ev_1, 0);
        cal_phase2_blk<<< 1, threads, 0, stream[i]>>>(p1_start, p2_start);
        hipStreamWaitEvent(stream[i], ev_2, 0);
        cal_phase3<<<Rounds, threads, 0, stream[i]>>>(p1_start, p2_start, 0);
    }
    //fprintf(stderr, "%d first round done\n", tid);

    for(int i=1; i<Rounds; i++) {
        p1_start += diag_size;

        cal_phase1<<<1, threads, 0, stream[i]>>>(p1_start);
        hipEventRecord(ev_1, stream[i]);
        cal_phase2_row<<<Rounds, threads, 0, stream[i]>>>(p1_start, i);
        hipEventRecord(ev_2, stream[i]);

        for(int j=0; j<Rounds; j++) {
            if(i==j) continue;

            p2_sub = p1_start + line_d * (j - i);
            //printf("\tp1=(%d,%d), p2=(%d,%d) stream %d\n", ROW_COL(p1_start), ROW_COL(p2_sub), j);
            hipStreamWaitEvent(stream[j], ev_1, 0);
            cal_phase2_blk<<< 1, threads, 0, stream[j]>>>(p1_start, p2_sub);
            hipStreamWaitEvent(stream[j], ev_2, 0);
            cal_phase3<<<Rounds, threads, 0, stream[j]>>>(p1_start, p2_sub, i);
        }
    }
    //fprintf(stderr, "%d all rounds done\n", tid);

    ptr_h = Dist_h;
    ptr_d = Dist_d;
    for(int i=0; i<Rounds; i++) {
        bline = i==Rounds-1 ? last_line : block_size;
        hipMemcpy2DAsync(ptr_h, n_bytes, ptr_d, pitch_bytes, n_bytes, bline, hipMemcpyDeviceToHost, stream[i]);
        ptr_h += line_n;
        ptr_d += line_d;
    }

    //gettimeofday(&temp_time, NULL);
    //printf("before output> %g s\n", CAL_TIME);
}
